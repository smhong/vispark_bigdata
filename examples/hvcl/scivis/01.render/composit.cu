
#include <hip/hip_runtime.h>

extern "C"{

    __global__ void image_pass(float* result, float *data, int block_idx, int dimx, int dimy, int scr_idx)
    {
	    int x = blockIdx.x * blockDim.x + threadIdx.x;
	    int y = blockIdx.y * blockDim.y + threadIdx.y;


        if (x < dimx && y < dimy)
        {
            result[y*dimx+x] = data[y*dimx + x]+1;
            //result[y*dimx+x] = 1;
        }
    }

    __global__ void composit(float* result, float *data, int num_data, int dimx, int dimy)
    {
	    int x = blockIdx.x * blockDim.x + threadIdx.x;
	    int y = blockIdx.y * blockDim.y + threadIdx.y;


        if (x < dimx && y < dimy)
        {
            float max_value = -1;
            
            for (int i = 0 ; i < num_data ; i++)
            {
                if (data[i*dimx*dimy + y*dimx + x] > max_value)
                {
                    max_value = data[i*dimx*dimy + y*dimx + x];
                }
            }
           
            result[y*dimx + x] = max_value;   
        }
    
    }
}
